#include "hip/hip_runtime.h"
#include "defns.cuh"

__global__ void my_print(hipTextureObject_t texObject, double * output) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  uint2 rval = tex1Dfetch<uint2>(texObject, idx);
  double dval = fetch_double(rval);
  __syncthreads();
  printf("TEX: %llu IDX: %d VAL: %0.16f\n", texObject, idx, dval);
  output[idx] = dval;
}

extern "C" void print_kernel_(hipTextureObject_t * obj, int * size) {
  // Kernel for calling my_print from Fortran
  double * d_out, * out;
  hipMalloc((void**)&d_out, sizeof(double) * (*size));
  out = (double*)malloc(sizeof(double) * (*size));
  
  // calls my_print from fortran
  my_print<<<*size,1>>>(*obj,d_out);
  hipMemcpy(out,d_out, sizeof(double) * (*size), hipMemcpyDeviceToHost);
  hipFree(d_out);
}

__global__ void fetch_idx(hipTextureObject_t texObject, int idx, double* out) {
  // Same as my_print, but yields a single value in device variable out
  uint2 rval = tex1Dfetch<uint2>(texObject, idx);
  *out = fetch_double(rval);
}

extern "C" void fetch_ (hipTextureObject_t * obj, int idx, double* out) {
  /* given a texture object obj, an index idx, put
   obj[idx] in the single double value out */

  double * d_out;
  hipMalloc((void**)&d_out, sizeof(double));
  fetch_idx<<<1,1>>>(*obj,idx,d_out);
  hipMemcpy(out,d_out, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_out);
}

__global__ void texturevals (hipTextureObject_t obj, double * output) {
  // fetches values in texture obj into gpu array output
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  uint2 rval = tex1Dfetch<uint2>(obj, idx);
  output[idx] = fetch_double(rval);
  __syncthreads();
 }

// kernel for texturevals
extern "C" void texturevals_kernel_
(hipTextureObject_t * obj, int size, double* out) {
  
  double * d_out;
  hipMalloc((void**)&d_out, size * sizeof(double));

  texturevals<<<size,1>>>(*obj,d_out);
  hipMemcpy(out,d_out, size * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_out);
}

extern "C" hipTextureObject_t setup_(double * data, int * size){
  // initialize device data
  
  double* d_data;
  hipMalloc((void**)&d_data,(*size)*sizeof(double));
  hipMemcpy(d_data, data, (*size)*sizeof(double), hipMemcpyHostToDevice);

  hipTextureDesc td;
  memset(&td, 0, sizeof(td));
  td.normalizedCoords = 0;
  td.addressMode[0] = hipAddressModeBorder;
  td.readMode = hipReadModeElementType;

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = d_data;
  resDesc.res.linear.sizeInBytes = *size*sizeof(double);
  resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
  resDesc.res.linear.desc.x = 32;
  resDesc.res.linear.desc.y = 32;

  hipTextureObject_t texObject;
  gpuErrchk(hipCreateTextureObject(&texObject, &resDesc, &td, NULL));

  // printf("LEAVING SETUP\n");  
  // printf("TEXID: %llu\n", texObject);
  // free(data);
  return texObject;
}

extern "C" void free_cuda_memory_() {
  hipDeviceReset();
}