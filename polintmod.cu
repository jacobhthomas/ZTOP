#include "hip/hip_runtime.h"
#include "defns.cuh"

__global__ void polint4f (double XA[4], double YA[4], double * X, double* Y) {
  double H1,H2,H3,H4,W,DEN,D1,C1,D2,C2,D3,C3,CD1,CC1,CD2,CC2,DD1,DC1;
  H1 = XA[0] - *X;
  H2 = XA[1] - *X;
  H3 = XA[2] - *X;
  H4 = XA[3] - *X;

  W   = YA[1] - YA[0];
  DEN = W / (H1 - H2);
  D1  = H2 * DEN;
  C1  = H1 * DEN;

  W   = YA[2] - YA[1];
  DEN = W/(H2-H3);
  D2  = H3*DEN;
  C2  = H2*DEN;

  W   = YA[3]-YA[2];
  DEN = W/(H3-H4);
  D3  = H4*DEN;
  C3  = H3*DEN;

  W   = C2-D1;
  DEN = W/(H1-H3);
  CD1 = H3*DEN;
  CC1 = H1*DEN;

  W   = C3-D2;
  DEN = W/(H2-H4);
  CD2 = H4*DEN;
  CC2 = H2*DEN;

  W   = CC2-CD1;
  DEN = W/(H1-H4);
  DD1 = H4*DEN;
  DC1 = H1*DEN;

  if(H3 + H4 < 0.0)
    *Y = YA[3] + D3 + CD2 + DD1;
  else if(H2 + H3 < 0.0)
    *Y = YA[2] + D2 + CD1 + DC1;
  else if(H1+H2 < 0.0)
    *Y = YA[1] + C2 + CD1 + DC1;
  else
    *Y = YA[0] + C1 + CC1 + DC1;
}

// Global device variables
double *device_XA, *device_YA, *device_X, *device_Y;

// gpu allocation
extern "C" void allocate_memory_(double XA[4], double YA[4], double* XV, double* YV) {
  hipMalloc((void**)device_X , sizeof(double));
  hipMalloc((void**)device_Y , sizeof(double));
  hipMalloc((void**)device_XA, sizeof(double) * 4);
  hipMalloc((void**)device_YA, sizeof(double) * 4);
}

// copy sent data to GPU
extern "C" void copy_memory_(double XA[4], double YA[4], double* XV, double* YV) {
  hipMemcpy(device_X , XV, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_XA, XA, sizeof(double) * 4, hipMemcpyHostToDevice);
  hipMemcpy(device_YA, YA, sizeof(double) * 4, hipMemcpyHostToDevice);
}

// gpu deallocation
extern "C" void free_memory_(double XA[4], double YA[4], double* XV, double* YV) {
  hipFree(device_X);
  hipFree(device_Y);
  hipFree(device_XA);
  hipFree(device_YA);
}

extern "C" void polint_wrapper_(double XA[4], double YA[4], double* XV, double* YV) {
  double * device_XA, * device_YA; // device vars
  double * device_X , * device_Y ;

  // allocate memory on device (GPU)
  hipMalloc((void**)&device_Y , sizeof(double));
  hipMalloc((void**)&device_X , sizeof(double));
  hipMalloc((void**)&device_XA, sizeof(double) * 4);
  hipMalloc((void**)&device_YA, sizeof(double) * 4);

  // copy sent data to GPU
  hipMemcpy(device_X , XV, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_XA, XA, sizeof(double) * 4, hipMemcpyHostToDevice);
  hipMemcpy(device_YA, YA, sizeof(double) * 4, hipMemcpyHostToDevice);

  // call on GPU
  polint4f<<<1,1>>>(device_XA, device_YA, device_X, device_Y);

  // copy value of Y to cpu
  hipMemcpy(YV, device_Y, sizeof(double), hipMemcpyDeviceToHost);

  // free memory
  hipFree(device_X);
  hipFree(device_Y);
  hipFree(device_XA);
  hipFree(device_YA);

}

extern "C" void polint_wrapper_modified_(double XA[4], double YA[4], double* XV, double* YV) {

  polint4f<<<1,1>>>(device_XA, device_YA, device_X, device_Y);

  hipMemcpy(YV, device_Y, sizeof(double), hipMemcpyDeviceToHost);

}