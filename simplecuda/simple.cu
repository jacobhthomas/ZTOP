
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void parentkern(double v[8])
{
  v[0] = 3.5;
}


extern "C" void simple_
(int * i1, double * v1, double vout[8])
{
  vout[2] = 2.3;
  double* device_v;
  hipMalloc((void**)&device_v, sizeof(double) * 8);
  
  parentkern<<<1,1>>>(device_v);
  hipDeviceSynchronize();
  hipMemcpy(vout, device_v, sizeof(double)*8, hipMemcpyDeviceToHost);

  hipFree(device_v);

  //  printf("%d\n",cudaGetLastError());
  printf("%s",hipGetErrorString(hipGetLastError()));
}

