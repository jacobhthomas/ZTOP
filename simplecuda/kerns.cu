
#include <hip/hip_runtime.h>
#include<stdio.h>

__device__ int ii1;

__global__ void childkern(int i1, double v1, double vout[8])
{
   int i = threadIdx.x;
   vout[i] = i;
}


__global__ void parentkern(int* i1, double* v1, double vout[8])
{
   ii1 = *i1;
   double vv1 = *v1;

   ii1 += 3;
//   childkern<<<1,8>>>(ii1, vv1, vout);
   vout[0] = 2.3;
      
}


extern "C" void kerns_
(int * i1, double * v1, double vout[8])
{
//  printf("%f\n",*vout[0]);
//  printf("%f\n",*vout[1]);

  int * device_i1;
  double * device_v1; // device vars
  double * device_vout ;

  // allocate memory on device (GPU)
  hipMalloc((void**)&device_i1 , sizeof(int));
  hipMalloc((void**)&device_v1 , sizeof(double));
  hipMalloc((void**)&device_vout, sizeof(double) * 8);

  // copy sent data to GPU
  hipMemcpy(device_i1 , i1, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_v1, v1, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_vout, vout, sizeof(double)*8, hipMemcpyHostToDevice);

  // call on GPU
  parentkern<<<1,1>>>(device_i1, device_v1, device_vout);

  // copy value of Y to cpu
  hipMemcpy(vout, device_vout, sizeof(double)*8, hipMemcpyDeviceToHost);
  // free memory
  hipFree(device_vout);
  hipFree(device_v1);
  hipFree(device_i1);
}
