#include "hip/hip_runtime.h"
/* Convert fortran function to cuda
   What variables do we need?
   Textures for X, T/Q, and PDF   
*/

#include "defns.cuh"

void bin_search (double * data, double val, int * lo, int * hi) {
  /* perform binary search to find an index bin pair [lo,hi] 
     in data closest to value of val */
   while(*hi - *lo > 1) {
     int mid = (*hi + *lo) / 2;
     if(val >= data[mid]) {
       *lo = mid;
     } else {
       *hi = mid;
    }
  }
}

void bin_search_check_x(int lo, int upper, int size, int * ret) {
  /* ensure that nothing went wrong when binary searching on x */
  if (lo <= -1) {
    // X cannot be less than '0'
    printf("X:Severe error in Binary Search! Lower than min\n");
    exit(20);
  } else if(lo == 0)
    // actual value is in first bin, closest to min
    *ret = 0;
  else if(lo <= size-2)
    // clip if high val
    *ret = lo - 1;
  else if(lo == size-1)
    *ret = lo - 2;
  else {
    // x is not bigger than 1
    printf("X:Severe error in Binary Search! Higher than max\n");
    exit(21);
  }
}

void bin_search_check_q(int lower, int upper, int size, int * ret) {
  /* ensure that nothing went wrong when binary searching on Q */
  if (lower <= 0) {
    *ret = 0;
  } else if(lower <= size-2) {
    *ret = lower - 1;
  } else {
    *ret = size - 3;
  }
}

extern "C" void partonx12_wrapper_
(int * iSetch, int *iParton, int * nx, int * nt, int * npts,
 int * NfMx, int * MxVal,
 double * XX, double* QQ, double* qB,
 hipTextureObject_t* xTex, hipTextureObject_t* tTex, hipTextureObject_t* pTex,
 double * ret) {

  // DECLARATIONS
  double X = *XX, Q = *QQ, qBase = *qB; 
  double svec1, svec2, svec3, svec4, s12, s13, s23, s24, s34, sy2, sy3,
    const1, const2, const3, const4, const5, const6, s1213, s2434, sdet, tmp,
    tvec1, tvec2, tvec3, tvec4, t12, t13, t23, t24, t34, ty2, ty3, tmp1, tmp2,
    tdet, *Fx, ss, tt;

  double Fij[4], Fvec[4];

  double * xv, * tv, * UPD, * xvpow, xpow = 0.3;

  int ip;
  // END DECLARATIONS
  
  // allocate size for x array, t array, and table array
  xv  = (double *) malloc(sizeof(double) * (*nx  ));
  tv  = (double *) malloc(sizeof(double) * (*nt  ));
  UPD = (double *) malloc(sizeof(double) * (*npts));

  // Place the values in the arrays from the textures
  texturevals_kernel_ (xTex, *nx  , xv); 
  texturevals_kernel_ (tTex, *nt  , tv);   
  texturevals_kernel_ (pTex, *npts, UPD);  

  // allocate xvpow array
  xvpow = (double*)malloc(sizeof(double) * (*nx));
  
  xvpow[0] = 0.0;
  // index at 0 
  for(int i = 0; i < *nx; i++) {
    xvpow[i] = pow(xv[i], xpow);
  }
    
  tt = log(log(Q/qBase));
  
  // binary search on x
  int lowerX = -1, upperX = *nx + 1, idxX;
  // search for most appropriate bin [lowerX, upperX] for x vals
  bin_search(xv, X, &lowerX, &upperX);
  // perform checks and find actual index value
  bin_search_check_x(lowerX, upperX, *nx, &idxX);

  ss = pow(X,xpow);
  
  if(lowerX >= 2 && lowerX <= *nx-2) {
    
    svec1 = xvpow[idxX];
    svec2 = xvpow[idxX+1];
    svec3 = xvpow[idxX+2];
    svec4 = xvpow[idxX+3];
    
    s12 = svec1 - svec2;
    s13 = svec1 - svec3;
    s23 = svec2 - svec3;
    s24 = svec2 - svec4;
    s34 = svec3 - svec4;
    
    sy2 = ss - svec2;
    sy3 = ss - svec3;
    
    const1 = s13/s23;
    const2 = s12/s23;
    const3 = s34/s23;
    const4 = s24/s23;
    s1213 = s12 + s13;
    s2434 = s24 + s34;
    sdet = s12*s34 - s1213*s2434;
    tmp = sy2*sy3/sdet;
    const5 = (s34*sy2-s2434*sy3)*tmp/s12;
    const6 = (s1213*sy2-s12*sy3)*tmp/s34;

  }
  
  // binary search on t (q)
  int lowerQ = -1, upperQ = *nt + 1, idxQ;
  bin_search(tv, tt, &lowerQ, &upperQ);
  bin_search_check_q(lowerQ, upperQ, *nt, &idxQ);

  if(lowerQ >= 1 && lowerQ <= *nt - 2) {
    tvec1 = tv[idxQ];
    tvec2 = tv[idxQ+1];
    tvec3 = tv[idxQ+2];
    tvec4 = tv[idxQ+3];
    
    t12 = tvec1 - tvec2;
    t13 = tvec1 - tvec3;
    t23 = tvec2 - tvec3;
    t24 = tvec2 - tvec4;
    t34 = tvec3 - tvec4;
    
    ty2 = tt - tvec2;
    ty3 = tt - tvec3;
    
    tmp1 = t12 + t13;
    tmp2 = t24 + t34;
    
    tdet = t12*t34 - tmp1*tmp2;
  }
  
  // done with first time setup

  if(*iParton > *MxVal) 
    ip = -*iParton;
  else
    ip = *iParton;

  int jtmp = ((ip + *NfMx)*(*nt+1)+(idxQ-1))*(*nx+1)+idxX+1;
  
  for(int i = 1; i <= nqvec; i++) {
    int tempIdx = jtmp + (i * (*nx+1));

    if(idxX == 0) {

      Fij[0] = 0;
      Fij[1] = UPD[tempIdx+1] * (pow(xv[1],2));
      Fij[2] = UPD[tempIdx+2] * (pow(xv[2],2));
      Fij[3] = UPD[tempIdx+3] * (pow(xv[3],2));

      polint_wrapper_(xvpow, Fij, &ss, Fx);

      if (X > 0.0)
	Fvec[i] = *Fx / pow(X,2.0);
      
    } else if(lowerX == *nx - 1) {

      polint_wrapper_(&xvpow[*nx-3], &UPD[tempIdx-1], &ss, Fx);
      Fvec[i] = *Fx;
      
    } else {
      
      double sf2, sf3, g1, g4;
      sf2 = UPD[tempIdx+0];
      sf3 = UPD[tempIdx+1];

      g1 =  sf2*const1 - sf3*const2;
      g4 = -sf2*const3 + sf3*const4;

      Fvec[i-1] = (const5 * (UPD[tempIdx-1]-g1)
	       + const6 * (UPD[tempIdx+2]-g4)
	       + sf2 * sy3 - sf3 * sy2) / s23;

    }
  }

  if(lowerQ <= 0) 
    polint_wrapper_(tv, Fvec, &tt, ret);
  else if (lowerQ >= *nt-1)
    polint_wrapper_(&tv[*nt-3], Fvec, &tt, ret);
  else {

    double tf2, tf3, g1,g4,h00;

    tf2 = Fvec[1];
    tf3 = Fvec[2];

    g1 = ( tf2*t13 - tf3*t12) / t23;
    g4 = (-tf2*t34 + tf3*t24) / t23;

    h00 = ((t34*ty2-tmp2*ty3)*(Fvec[0]-g1)/t12 + (tmp1*ty2-t12*ty3)*(Fvec[3]-g4)/t34);

    *ret = (h00*ty2*ty3/tdet + tf2*ty3 - tf3*ty2) / t23;
    
  }
  
  free(xv); free(tv); free(UPD);
  return;
  
}


